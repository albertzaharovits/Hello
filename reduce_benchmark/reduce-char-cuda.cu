#include "hip/hip_runtime.h"
// CUDA reduce kernel 
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include <sys/time.h>

#define SAFE_LOAD_GLOBAL(s, i, c) \
	((size_t)(i)) < ((size_t)(c)) ? \
			((const char*)(s))[(size_t)(i)] : SCHAR_MIN

#define GET_TIME_DELTA(t1, t2) (((t2).tv_sec - (t1).tv_sec) * 1000.0 + \
					((t2).tv_usec - (t1).tv_usec) / 1000.0)

typedef unsigned int uint;

__device__
void warpReduce(char *d_data, uint localIdx, uint groupDim)
{
		if(localIdx < 32)
		{
			if(groupDim >= 64)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+32]);
			if(groupDim >= 32)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+16]);
			if(groupDim >= 16)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+8]);
			if(groupDim >= 8)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+4]);
			if(groupDim >= 4)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+2]);
			if(groupDim >= 2)
				d_data[localIdx] = max(d_data[localIdx], d_data[localIdx+1]);
		}
}

__global__
void find_highest_ascii(const char *g_idata, uint count, char* g_odata)
{
	extern __shared__ char d_data[];
	char local_max = SCHAR_MIN;
	uint global_size = gridDim.x*blockDim.x;
	uint stride = (count + global_size-1)/global_size;
	uint globalIdx = threadIdx.x + blockIdx.x*(stride*blockDim.x);

	for(uint s=0;s<stride;s++) {
		local_max = max(local_max, (char)(SAFE_LOAD_GLOBAL(g_idata, 
			globalIdx, count)));
		globalIdx += blockDim.x;
	}

	d_data[threadIdx.x] = local_max;
	__syncthreads();

	for(uint s = blockDim.x >> 1;s>32;s>>=1)
	{
		if(threadIdx.x < s)
			d_data[threadIdx.x] = max(d_data[threadIdx.x],d_data[threadIdx.x+s]);

		__syncthreads();
	}

	warpReduce(d_data, threadIdx.x, blockDim.x);

	if(threadIdx.x == 0)
		g_odata[blockIdx.x] = d_data[0];

}

__global__
void find_highest_ascii2(const char *g_idata, uint count, char* g_odata)
{
	extern __shared__ char d_data[];
	char local_max = SCHAR_MIN;
	uint global_size = gridDim.x*blockDim.x;
	uint stride = (count + global_size-1)/global_size;
	uint globalIdx = threadIdx.x + blockIdx.x*(stride*blockDim.x);

	for(uint s=0;s<stride;s++) {
		local_max = max(local_max, (char)(SAFE_LOAD_GLOBAL(g_idata, 
			globalIdx, count)));
		globalIdx += blockDim.x;
	}

	d_data[threadIdx.x] = local_max;
	__syncthreads();

	for(uint s = blockDim.x >> 1;s>=1;s>>=1)
	{
		if(threadIdx.x < s)
			d_data[threadIdx.x] = max(d_data[threadIdx.x],d_data[threadIdx.x+s]);

		__syncthreads();
	}

	if(threadIdx.x == 0)
		g_odata[blockIdx.x] = d_data[0];
}

int main(int argc, char** argv)
{
	struct timeval t1, t2;
	uint size = 1<<28;
	uint temp_buffer_size = 1<<14;
	uint i;
	char *d_char_buffer, *d_temp_buffer, *d_result_buffer;
	char *d_char_buffer2, *d_temp_buffer2, *d_result_buffer2;
	char result_char;
	hipError_t error;

	char *host_input = (char*)malloc(size*sizeof(char));
	for(i=0;i<size;++i)
		host_input[i] = 'A' + i%20;

	host_input[3*(size/7)] = 'Z';

	printf("with/without loop unroll\n");
	/*****************WITH LOOP UNROLL*********************************/
	size = 1<<28;
	error = hipMalloc((void **)&d_char_buffer, size*sizeof(char));
	checkCudaErrors(error);
	error = hipMalloc((void **)&d_temp_buffer, temp_buffer_size*sizeof(char));
	checkCudaErrors(error);
	error = hipMalloc((void **)&d_result_buffer, sizeof(char));
	checkCudaErrors(error);
	error = hipMemcpy(d_char_buffer, host_input, size*sizeof(char),
		hipMemcpyHostToDevice);
	checkCudaErrors(error);

	gettimeofday(&t1, NULL);
	find_highest_ascii<<<(1<<14),(1<<10),(1<<10)*sizeof(char)>>>
		(d_char_buffer, size, d_temp_buffer);
	size = 1<<14;
	hipDeviceSynchronize();checkCudaErrors(hipGetLastError());
	find_highest_ascii<<<1,(1<<10),(1<<10)*sizeof(char)>>>
		(d_temp_buffer, size, d_result_buffer);
	hipDeviceSynchronize();checkCudaErrors(hipGetLastError());
	gettimeofday(&t2, NULL);

	error = hipMemcpy(&result_char, d_result_buffer, sizeof(char),
		hipMemcpyDeviceToHost);
	checkCudaErrors(error);
	assert(result_char == 'Z');
	printf("%lf ", GET_TIME_DELTA(t1,t2));
	hipFree(d_char_buffer);
	hipFree(d_temp_buffer);
	hipFree(d_result_buffer);
	
	/*****************WITHOUT LOOP UNROLL*********************************/
	size = 1<<28;
	error = hipMalloc((void **)&d_char_buffer2, size*sizeof(char));
	checkCudaErrors(error);
	error = hipMalloc((void **)&d_temp_buffer2, temp_buffer_size*sizeof(char));
	checkCudaErrors(error);
	error = hipMalloc((void **)&d_result_buffer2, sizeof(char));
	checkCudaErrors(error);
	error = hipMemcpy(d_char_buffer2, host_input, size*sizeof(char),
		hipMemcpyHostToDevice);
	checkCudaErrors(error);

	// no loop unrolled kernel
	gettimeofday(&t1, NULL);
	find_highest_ascii2<<<(1<<14),(1<<10),(1<<10)*sizeof(char)>>>
		(d_char_buffer2, size, d_temp_buffer2);
	size = 1<<14;
	find_highest_ascii2<<<1,(1<<10),(1<<10)*sizeof(char)>>>
		(d_temp_buffer2, size, d_result_buffer2);
	hipDeviceSynchronize();checkCudaErrors(hipGetLastError());
	gettimeofday(&t2, NULL);

	error = hipMemcpy(&result_char, d_result_buffer2, sizeof(char),
		hipMemcpyDeviceToHost);
	checkCudaErrors(error);
	assert(result_char == 'Z');
	printf("%lf\n", GET_TIME_DELTA(t1,t2));

	hipFree(d_char_buffer2);
	hipFree(d_temp_buffer2);
	hipFree(d_result_buffer2);
	return 0;
}

